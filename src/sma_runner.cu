#include <iostream>
#include "../include/sma_runner.cuh"
#include "../include/sma_cuda_func.cuh"

void wa_runner_cuda(struct algorithm_data& AD, struct timing_data& TD) {
    // Setup timing information
    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    // Initialize cublas (cuda linear algebra library)
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    std::cout << "hipblasCreate: " << cublasGetStatusString(status) << std::endl;

    // Define pointer for cuda avg_vector
    float* cudaV;

    // Create vector for matrix reduction operation
    float* avg_vector;
    avg_vector = new float[WINDOW_SIZE];
    for (size_t i = 0; i < WINDOW_SIZE; i++) {avg_vector[i] = (float)1 / WINDOW_SIZE;}

    // Allocate memory for raw data, window average array and avg_vector
    hipMalloc(&AD.cudaRD, AD.NUM_VALUES * sizeof(float));
    hipMalloc(&AD.cudaWA, AD.NUM_WINDOWS * sizeof(float));
    hipMalloc(&cudaV, WINDOW_SIZE * sizeof(float));

    // Transfer raw_data array and no. windows to GPU memory
    hipMemcpy(AD.cudaRD, AD.raw_data, AD.NUM_VALUES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaV, avg_vector, WINDOW_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Run cublas float gemv with a thread count of WINDOW_SIZE in each block
    float alpha = 1.0;
    float beta = 0.0;
    status = hipblasSgemv(handle, HIPBLAS_OP_T, WINDOW_SIZE, AD.NUM_WINDOWS, &alpha, AD.cudaRD, WINDOW_SIZE, cudaV, 1, &beta, AD.cudaWA, 1);
    std::cout << "hipblasSgemv: " << cublasGetStatusString(status) << std::endl;

    // Free allocated memory
    hipFree(cudaV);
    status = hipblasDestroy(handle);
    std::cout << "hipblasDestroy: " << cublasGetStatusString(status) << std::endl;

    // Record timing information
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float   elapsed_time;
    hipEventElapsedTime( &elapsed_time, start, stop );
    printf( "Time to window average:  %3.4f ms\n", elapsed_time );
    TD.avg_delta_wavg += elapsed_time;
}

void find_peaks_cuda_runner(struct algorithm_data& AD, struct timing_data& TD) {
    // Setup timing information
    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    // Local copy of no. windows as int to pass to cuda kernal
    int NUM_WINDOWS = static_cast<int>(AD.NUM_WINDOWS);

    // Allocate memory for maxima and no. windows
    hipMalloc(&AD.cudaM, (AD.NUM_WINDOWS - 2) * sizeof(float));
    hipMalloc(&AD.cudaNUM_WINDOWS, sizeof(int));

    // Transfer no. windows to GPU memory
    hipMemcpy(AD.cudaNUM_WINDOWS, &NUM_WINDOWS, sizeof(int), hipMemcpyHostToDevice);

    // Run find_peaks_cuda func. with a thread count of 20 in each block
    find_peaks_cuda <<< (AD.NUM_WINDOWS / 20), 20 >>> (AD.cudaWA, AD.cudaM, AD.cudaNUM_WINDOWS);

    // Free allocated memory
    hipFree(AD.cudaWA);
    hipFree(AD.cudaNUM_WINDOWS);

    // Record timing information
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float   elapsed_time;
    hipEventElapsedTime( &elapsed_time, start, stop );
    printf( "Time to find peaks:  %3.4f ms\n", elapsed_time );
    TD.avg_delta_peak += elapsed_time;
}

void find_minima_cuda_runner(struct algorithm_data& AD, struct timing_data& TD) {
    // Setup timing information
    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );
    
    // Allocate memory on GPU for minima data
    hipMalloc(&AD.cudaMI, AD.NUM_WINDOWS * sizeof(float));

    // Run find_minima_cuda
    find_minima_cuda <<< (AD.NUM_WINDOWS / 20), 20 >>> (AD.cudaRD, AD.cudaM, AD.cudaMI);

    // Copy wa back to gpu memory
    hipMemcpy(AD.minima, AD.cudaMI, AD.NUM_WINDOWS * sizeof(float), hipMemcpyDeviceToHost);

    // Free allocated memory
    hipFree(AD.cudaM);
    hipFree(AD.cudaRD);
    hipFree(AD.cudaMI);

    // Record timing information
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float   elapsed_time;
    hipEventElapsedTime( &elapsed_time, start, stop );
    printf( "Time to find minima:  %3.4f ms\n", elapsed_time );
    TD.avg_delta_min += elapsed_time;
}

void sma(struct algorithm_data& AD, struct timing_data& TD) {
    // Allocate memory to store window average data
    AD.window_average_data = new float[AD.NUM_VALUES / WINDOW_SIZE];

    // Allocate memory to store maxima data
    AD.maxima = new float[AD.NUM_VALUES / WINDOW_SIZE];
    AD.maxima[0] = 0;
    AD.maxima[AD.NUM_VALUES / WINDOW_SIZE - 1] = 0;

    // Allocate memory to store minima
    AD.minima = new float[AD.NUM_WINDOWS];

    // Run window averaging
    wa_runner_cuda(AD, TD);

    // Run find peaks to locate pulses
    find_peaks_cuda_runner(AD, TD);

    // Run find minima to find minimum in pulse containing regions
    find_minima_cuda_runner(AD, TD);

    /*
    // Count found minima to ensure correct value
    for (size_t i = 0; i < AD.NUM_WINDOWS; i++) {
        if (AD.minima[i] != 0) {
            AD.minima_count += 1;
        }
    }
    std::cout << "Minima found: " << AD.minima_count << std::endl;
    */

    // Delete allocated memory for window average data and maxima
    delete[] AD.window_average_data;
    delete[] AD.maxima;
}